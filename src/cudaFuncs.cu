#include "hip/hip_runtime.h"

//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////
///////   This file contains cuda functions Implementation   /////////////////
//////////////////////////////////////////////////////////////////////////////
#include "cudaFuncs.h"
#include <hipsolver.h>
#include <hiprand.h>
#include "math.h"

//This function does the cholesky decomposition
/* 
  input parameters: inMat: input matrix
                    dim: dimension of matrix
                    uplo: matrix fill type
*/
void chol(double* inMat, int dim, hipblasFillMode_t uplo ){×
   //variables for cuSolver cholesky 
   hipsolverHandle_t csrHandle = NULL;
   hipsolverStatus_t status;[MaÔ[MaÔ[MaÔ[MaÔ
   bool TESTFLAG = false;
   //variables for workspace
   int workSize = 0;[MaÓ[MaÓ[MaÓ
   double* workPtr;

   int* devInfo; //used for error checking

   //allocate shared memory
   hipMallocManaged( &devInfo, sizeof(int) );
   //create handle for library
   status = hipsolverDnCreate( &csrHandle );
   //get buffer size[MaÓ[MaÓ
   status = hipsolverDnDpotrf_bufferSize(csrHandle, uplo, dim, 
                                                   inMat, dim, &workSize );
   assert( status == HIPSOLVER_STATUS_SUCCESS );
   //allocate memory for workspace
   hipMallocManaged( &workPtr, workSize * sizeof(double) );
   
   //print starting matrix for error checking
   if(TESTFLAG){
     printf("\n Matrix before decomp: \n");
     for(int i = 0; i < dim; i++ ){
        for(int j = 0; j < dim; j++ ){
           printf(" %f", inM[MaÓ[MaÓ[MaÓ[MaÓat[ i*dim + j ]);
        }
        printf("\n");
     }[MaÓ[MaÓ[MaÓ
   }   

   //This step calls the cholesky function from cuSolver
     /* Function parameters: 
     hipsolverHandle_t: handle to cuSolver library
     hipblasFillMode_t: Indicates of matrix A lower or upper part stored
     int: dimension of matrix A
     float*: pointer to input matrix
     int: leading [MaÓ[MaÓdimension of 2D array used to store matrix
     float*:workspace pointer
     int: size of workspace
     int*: return for erro[MaÓr checking
     */ 
  status = hipsolverDnDpotrf(csrHandle, uplo, dim, inMat, dim, 
                                          workPtr, workSize, devInfo);
  
   assert( status == HIPSOLVER_STATUS_SUCCESS );  

  printf("\n Dev Info for cholesky: %d", *devInfo);
  
  if(TESTFLAG){
    //print final results for checking
    printf("\n Matrix after decomp: \n");
     for(int i = 0; i < dim; i++ ){
        for(int j = 0; j < dim; j++ ){
           printf(" %f", inMat[ i*dim + j ]);
        }
        printf("\n");
     }
  } 

  //destroy cusolver handle
  status = hipsolverDnDestroy( csrHandle );
  assert( status == HIPSOLVER_STATUS_SUCCESS );
}


//This function generates pseudo random standard normal distribution
void normGen( double* outputPtr, size_t n, double mean, double stddev, int seed ){
   //declare variables
   hiprandGenerator_t randHandle;
   hiprandStatus_t status;
   //First must create generator and set options
   status = hiprandCreateGenerator( &randHandle, HIPRAND_RNG_PSEUDO_DEFAULT );
   assert( status == HIPRAND_STATUS_SUCCESS && "create generator");
   //This step calls the random number generator function from cuRand
   /* Function paramters:
      hiprandGenerator_t : handle to generator
      float * outputPtr : pointer to array storing numbers
      size_t num        : 
      double mean       : Given mean
      double stddev     : Given standard deviation
   */
   status = hiprandSetPseudoRandomGeneratorSeed( randHandle, seed );
   assert( status == HIPRAND_STATUS_SUCCESS && "seeder");

   status = hiprandGenerateNormalDouble( randHandle, outputPtr, n, mean, stddev );
   assert( status == HIPRAND_STATUS_SUCCESS && "hiprand function call");

   //print results for testing purposes
   /*printf("\n Psuedo random standard normal matrix: \n");
   for(int i = 0; i < 3; i++ ){
      for(int j = 0; j < 3; j++ ){
         printf(" %f", outputPtr[ i*n + j ]);
      }
      printf("\n");
   }*/
   status = hiprandDestroyGenerator( randHandle );
   assert( status == HIPRAND_STATUS_SUCCESS && "destroyer" );
}

//square matrix mult
//C = alpha*op(A)op(B) + beta*C
void matMult( double* matA, double* matB, double* outMat, int dim ){
  //declare variables
  hipblasHandle_t myHandle;
  hipblasStatus_t status;
  double zero = 0;
  double one = 1;
  //variables for if matrix is normal, transpose, or hermitian t.
  hipblasOperation_t transa = HIPBLAS_OP_N;
  hipblasOperation_t transb = HIPBLAS_OP_N;

  //create library instance
  status = hipblasCreate( &myHandle );
  assert( status == HIPBLAS_STATUS_SUCCESS );

  status = hipblasDgemm( myHandle, transa, transb, dim, dim, dim, &one, matB,
                                            dim, matA, dim, &zero, outMat, dim );

  assert( status == HIPBLAS_STATUS_SUCCESS );

  //destroy cublas instance
  hipblasDestroy( myHandle );
}

//inverse CDF function, calls device function normcdfinv from CUDA math API
__global__ void invCDF( double* inMat, int n ){
  //initialzie variables for block id and thread id
    int bidx, tid;  
    bidx = blockIdx.x;
    tid = threadIdx.x;

    //grid stride loop
    for( int i = blockIdx.x * blockDim.x + threadIdx.x;  i < n; i+= blockDim.x * gridDim.x ){
       inMat[i] = normcdfinv( inMat[i] ); 
          
    }




}
