#include "hip/hip_runtime.h"

//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////
///////   This file contains cuda functions Implementation   /////////////////
//////////////////////////////////////////////////////////////////////////////
#include "cudaFuncs.h"
#include <hipsolver.h>
#include <hiprand.h>
#include "math.h"
#include "stats.hpp"

//This function does the cholesky decomposition
/* 
  input parameters: inMat: input matrix
                    dim: dimension of matrix
                    uplo: matrix fill type
*/
void chol(double* inMat, int dim, hipblasFillMode_t uplo ){
   //variables for cuSolver cholesky 
   hipsolverHandle_t csrHandle = NULL;
   hipsolverStatus_t status;
   bool TESTFLAG = true;
   //variables for workspace
   int workSize = 0;
   double* workPtr;

   int* devInfo; //used for error checking

   //allocate shared memory
   hipMallocManaged( &devInfo, sizeof(int) );
   //create handle for library
   status = hipsolverDnCreate( &csrHandle );
   //get buffer size
   status = hipsolverDnDpotrf_bufferSize(csrHandle, uplo, dim, 
                                                   inMat, dim, &workSize );
   assert( status == HIPSOLVER_STATUS_SUCCESS );
   //allocate memory for workspace
   hipMallocManaged( &workPtr, workSize * sizeof(double) );
   
   //print starting matrix for error checking
   if(TESTFLAG){
     printf("\n Matrix before decomp: \n");
     for(int i = 0; i < dim; i++ ){
        for(int j = 0; j < dim; j++ ){
           printf(" %f", inMat[ i*dim + j ]);
        }
        printf("\n");
     }
   }   

   //This step calls the cholesky function from cuSolver
     /* Function parameters: 
     hipsolverHandle_t: handle to cuSolver library
     hipblasFillMode_t: Indicates of matrix A lower or upper part stored
     int: dimension of matrix A
     float*: pointer to input matrix
     int: leading dimension of 2D array used to store matrix
     float*:workspace pointer
     int: size of workspace
     int*: return for error checking
     */ 
  status = hipsolverDnDpotrf(csrHandle, uplo, dim, inMat, dim, 
                                          workPtr, workSize, devInfo);
  
   assert( status == HIPSOLVER_STATUS_SUCCESS );  

  printf("\n Dev Info for cholesky: %d", *devInfo);
  
  if(TESTFLAG){
    //print final results for checking
    printf("\n Matrix after decomp: \n");
     for(int i = 0; i < dim; i++ ){
        for(int j = 0; j < dim; j++ ){
           printf(" %f", inMat[ i*dim + j ]);
        }
        printf("\n");
     }
  } 

  //destroy cusolver handle
  status = hipsolverDnDestroy( csrHandle );
  assert( status == HIPSOLVER_STATUS_SUCCESS );
}


//This function generates pseudo random standard normal distribution
void normGen( double* outputPtr, size_t n, double mean, double stddev, int seed ){
   //declare variables
   hiprandGenerator_t randHandle;
   hiprandStatus_t status;
   //First must create generator and set options
   status = hiprandCreateGenerator( &randHandle, HIPRAND_RNG_PSEUDO_DEFAULT );
   assert( status == HIPRAND_STATUS_SUCCESS && "create generator");
   //This step calls the random number generator function from cuRand
   /* Function paramters:
      hiprandGenerator_t : handle to generator
      float * outputPtr : pointer to array storing numbers
      size_t num        : 
      double mean       : Given mean
      double stddev     : Given standard deviation
   */
   status = hiprandSetPseudoRandomGeneratorSeed( randHandle, seed );
   assert( status == HIPRAND_STATUS_SUCCESS && "seeder");

   status = hiprandGenerateNormalDouble( randHandle, outputPtr, n, mean, stddev );
   assert( status == HIPRAND_STATUS_SUCCESS && "hiprand function call");

   //print results for testing purposes
   /*printf("\n Psuedo random standard normal matrix: \n");
   for(int i = 0; i < 3; i++ ){
      for(int j = 0; j < 3; j++ ){
         printf(" %f", outputPtr[ i*n + j ]);
      }
      printf("\n");
   }*/
   status = hiprandDestroyGenerator( randHandle );
   assert( status == HIPRAND_STATUS_SUCCESS && "destroyer" );
}

//square matrix mult
//C = alpha*op(A)op(B) + beta*C
void matMult( double* matA, double* matB, double* outMat, int dim ){
  //declare variables
  hipblasHandle_t myHandle;
  hipblasStatus_t status;
  double zero = 0;
  double one = 1;
  //variables for if matrix is normal, transpose, or hermitian t.
  hipblasOperation_t transa = HIPBLAS_OP_N;
  hipblasOperation_t transb = HIPBLAS_OP_N;

  //create library instance
  status = hipblasCreate( &myHandle );
  assert( status == HIPBLAS_STATUS_SUCCESS );

  status = hipblasDgemm( myHandle, transa, transb, dim, dim, dim, &one, matB,
                                            dim, matA, dim, &zero, outMat, dim );

  assert( status == HIPBLAS_STATUS_SUCCESS );

  //destroy cublas instance
  hipblasDestroy( myHandle );
}

//This device function calculates the normal cdf of the input
// calls normcdf from CUDA math API
__global__ void normCDF( double* inMat, int n ){
  //initialzie variables for block id and thread id
    //int bidx, tid;  
    //bidx = blockIdx.x;
    //tid = threadIdx.x;

    //grid stride loop
    for( int i = blockIdx.x * blockDim.x + threadIdx.x;  i < n; i+= blockDim.x * gridDim.x ){
      //printf("HELLO FROM INVCDF loop! \n");
       
       inMat[i] = normcdf( inMat[i] ); 
       printf("%f ", inMat[i] );
          
    }

}

//function for testing calling stat library from kernel
__global__ void testFunc( double* inMat, int cols ){
  //int bidx, tid;
  double temp;

  //temp = stats::qnorm( inMat[ blockIdx.x * blockDim.x + threadIdx.x ] );

  //grid stride
  for( int i = blockIdx.x * blockDim.x + threadIdx.x; i < cols; i+= blockDim.x*gridDim.x ){
    
    inMat[i] = stats::qnorm( inMat[i] );
    printf("%f ", inMat[i] );
  }
  
}
