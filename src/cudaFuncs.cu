
//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////
///////   This file contains cuda functions Implementation   /////////////////
//////////////////////////////////////////////////////////////////////////////
#include "cudaFuncs.h"
#include <hipsolver.h>
#include <hiprand.h>

void chol(double* inMat, int dim, hipblasFillMode_t uplo ){
   //variables for cuSolver cholesky 
   hipsolverHandle_t csrHandle = NULL;
   hipsolverStatus_t status;
   
   //variables for workspace
   int workSize = 0;
   double* workPtr;

   int* devInfo; //used for error checking

   //allocate shared memory
   hipMallocManaged( &devInfo, sizeof(int) );
   //create handle for library
   status = hipsolverDnCreate( &csrHandle );
   //get buffer size
   status = hipsolverDnDpotrf_bufferSize(csrHandle, uplo, dim, 
                                                   inMat, dim, &workSize );
   assert( status == HIPSOLVER_STATUS_SUCCESS );
   //allocate memory for workspace
   hipMallocManaged( &workPtr, workSize * sizeof(double) );
   //print starting matrix for error checking
   printf("\n Matrix before decomp: \n");
   for(int i = 0; i < dim; i++ ){
      for(int j = 0; j < dim; j++ ){
         printf(" %f", inMat[ i*dim + j ]);
      }
      printf("\n");
   }
      

   //This step calls the cholesky function from cuSolver
     /* Function parameters: 
     hipsolverHandle_t: handle to cuSolver library
     hipblasFillMode_t: Indicates of matrix A lower or upper part stored
     int: dimension of matrix A
     float*: pointer to input matrix
     int: leading dimension of 2D array used to store matrix
     float*:workspace pointer
     int: size of workspace
     int*: return for error checking
     */ 
  status = hipsolverDnDpotrf(csrHandle, uplo, dim, inMat, dim, 
                                          workPtr, workSize, devInfo);
  
   assert( status == HIPSOLVER_STATUS_SUCCESS );  

  printf("\n Dev Info for cholesky: %d", *devInfo);
  
  //print final results for checking
  printf("\n Matrix after decomp: \n");
   for(int i = 0; i < dim; i++ ){
      for(int j = 0; j < dim; j++ ){
         printf(" %f", inMat[ i*dim + j ]);
      }
      printf("\n");
   }
}

//This function generates pseudo random standard normal distribution
void normGen( double* outputPtr, int n ){
   //declare variables
   hiprandGenerator_t randHandle;
   hiprandStatus_t status;
   //First must create generator and set options
   status = hiprandCreateGenerator( &randHandle, HIPRAND_RNG_PSEUDO_XORWOW );
   assert( status == HIPRAND_STATUS_SUCCESS );
   //This step calls the random number generator function from cuRand
   /* Function paramters:
      hiprandGenerator_t : handle to generator
      float * outputPtr : pointer to array storing numbers
      size_t num        : 
      float mean        : Given mean
      float stddev      : Given standard deviation
   */
   status = hiprandGenerateNormalDouble( randHandle, outputPtr, n, 0, 0 );
   assert( status == HIPRAND_STATUS_SUCCESS );

   //print results for testing purposes
   printf("\n Psuedo random standard normal matrix: \n");
   for(int i = 0; i < n; i++ ){
      for(int j = 0; j < n; j++ ){
         printf(" %f", outputPtr[ i*n + j ]);
      }
      printf("\n");
   }
}
