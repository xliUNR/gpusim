//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////
///////   This file contains cuda functions Implementation   /////////////////
//////////////////////////////////////////////////////////////////////////////
#include "cudaFuncs.h"
#include <hipsolver.h>
#include 

void chol(int* inMat, int dim, hipblasFillMode_t uplo ){
   //variables for cuSolver cholesky 
   hipsolverHandle_t csrHandle = NULL;
   hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;
   hipsolverStatus_t status;
   
   //variables for workspace
   int workSize = 0;
   double* workPtr;

   int* devInfo //used for error checking

   //allocate shared memory
   hipMallocManaged( &devInfo, sizeof(int) );
   //create handle for library
   status = hipsolverDnCreate( &csrHandle );
   //get buffer size
   status = cussolverDnDpotrf_bufferSize(csrHandle, uplo, dim, 
                                                   inMat, dim, &workSize );
   assert( status == HIPSOLVER_STATUS_SUCCESS );
   //allocate memory for workspace
   hipMallocManaged( &workPtr, workSize * sizeof(double) );
   //print starting matrix for error checking
   printf("\n Matrix before decomp: ");
   for(int i = 0; i < dim; i++ ){
      for(int j = 0; j < dim, j++ ){
         printf(" %f", inMat[ i*dim + j ]);
      }
      printf("\n");
   }
      

   //This step calls the cholesky function from cuSolver
     /* Function parameters: 
     hipsolverHandle_t: handle to cuSolver library
     hipblasFillMode_t: Indicates of matrix A lower or upper part stored
     int: dimension of matrix A
     float*: pointer to input matrix
     int: leading dimension of 2D array used to store matrix
     float*:workspace pointer
     int: size of workspace
     int*: return for error checking
     */ 
  status = hipsolverDnDpotrf(csrHandle, uplo, dim, inMat, dim, 
                                          workPtr, workSize, devInfo);
  
   assert( status == HIPSOLVER_STATUS_SUCCESS );  

  printf("\n Dev Info for cholesky: %d", *devInfo);
  
  //print final results for checking
  printf("\n Matrix after decomp: ");
   for(int i = 0; i < dim; i++ ){
      for(int j = 0; j < dim, j++ ){
         printf(" %f", inMat[ i*dim + j ]);
      }
      printf("\n");
   }
}

//This function generates normal distributed
void normGen()