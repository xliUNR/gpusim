///////////////////////////////////////////////////////////////////////////////
///////////////   This is the GPU version of NORTA   //////////////////////////
///////////////////// Written by Eric Li //////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////



/////////////////////////////  Includes  //////////////////////////////////////
#include <cstdlib>
#include <fstream>
#include <cstdio>
#include <iostream>
#include <stdlib.h>
#include <hipsolver.h>
#include <assert.h>
#include <hip/hip_runtime.h>
//#include <hiprand.h>
#include "cudaFuncs.h"

using namespace std;






///////////////////////////////////////////////////////////////////////////////
/////////////////////////////////    Main   ///////////////////////////////////
int main( int argc, char const *argv[])
{
 //initialize variables
   //initiliaze arrays for holding input data
   double* r20Arr; 
   //double* r200Arr;
   int r20n = 20;

   int r20Size; 
   //int r200Size;
   ifstream srcFile;
   float a;
   double A0[3*3] = { 1.0, 2.0, 3.0, 2.0, 5.0, 5.0, 3.0, 5.0, 12.0 };
   double* dA0;
   double* sim_data;

   //r200n = 200;
   r20Size = r20n*r20n;
   //r200Size = r200n*r200n;
   //cuSolver 
   //cuSolverStatus_t solverStatus;
   
   //hipStream_t stream = NULL;   
   
   //set stream
   //hipsolverSetStream(csrHandle, stream);

   //print cusolver version
   int major=-1,minor=-1,patch=-1;
    cusolverGetProperty(MAJOR_VERSION, &major);
    cusolverGetProperty(MINOR_VERSION, &minor);
    cusolverGetProperty(PATCH_LEVEL, &patch);
    printf("CUSOLVER Version (Major,Minor,PatchLevel): %d.%d.%d\n", major,minor,patch);

   //allocated unified memory for storage of input covar matrix. 
   hipMallocManaged(&r20Arr, r20Size*sizeof(double));
   
   //allocate device memory for simple testing
   hipMallocManaged( &dA0, 9*sizeof(double) );
   
   //copy explicitly defined matrix into device
   hipMemcpy( dA0, A0, 9*sizeof(double), hipMemcpyHostToDevice );
   //hipMallocManaged(&r200Arr, r200Size*sizeof(float));
     
   //Timing for file read
   hipEvent_t read readStart, readEnd;
   hipEventCreate( &readStart, &readEnd );
   hipEventRecord( readStart, 0); 

   //Section for reading in arrays from file
   srcFile.open("../test_corr_matrix_d=20.txt", fstream::in);
   if(srcFile)
      {
        cout << endl << "SUCCESSFUL FILE OPEN";
	 for(int i = 0; i < r20Size; i++)
          {
            srcFile >> a;
	          //cout << a << "|";
            if( !(i % 20) ){ cout << endl;}

            r20Arr[i] = a;
          } 
          
      }
    else
      {
        cout << std::endl << "ERROR OPENING FILE";
      }

 //close file
 srcFile.close();

 //stop timing
 hipEventRecord( readEnd, 0 );
 hipEventSynchronize( readEnd );
 float readTime;
 hipEventElapsedTime( &readTime
 //test input read by printing results
  printf("\n INITIAL MATRIX\n");
 
  for(int i = 0; i < 3; i++ ){
    for(int j = 0; j <3; j++ )
      {
        printf(" %f", dA0[i*3+j]);
      } 
      printf("\n");
   } 
//cholesky decomp with floats (specified by S)
/*  //initialize variables
  hipsolverHandle_t csrHandle = NULL;
  hipblasFillMode_t uplo= HIPBLAS_FILL_MODE_UPPER;
  hipsolverStatus_t status;
  int r20workSize = 0;
  double* r20work;
  int* devInfo; //used for error checking
  
  hipMallocManaged(&devInfo, sizeof(int));
  //double* r200work;
  //create cusolver handle 
  status = hipsolverDnCreate(&csrHandle);
  assert(HIPSOLVER_STATUS_SUCCESS == status);

  //This is the Cholesky decomp step 
  //First calculate size of workspace
  */
  //float r200work;
  /*status = hipsolverDnDpotrf_bufferSize(csrHandle, 
                                uplo, r20n, r20Arr, r20n, &r20workSize);
  status = hipsolverDnDpotrf_bufferSize(csrHandle, uplo, 3, dA0, 3, &r20workSize);
  assert(HIPSOLVER_STATUS_SUCCESS == status );

  //hipsolverDnSpotrf_bufferSize(csrHandle, 
  //                              uplo, r200n, r200Arr, r200n, r200workSize);
 
  //Allocate memory for workspace
  hipMallocManaged( &r20work, r20workSize*sizeof(double) );
  //hipMallocManaged(&r200work, r200workSize*sizeof(float));
  
  //This step calls the cholesky function from cuSolver
     Function parameters: 
     hipsolverHandle_t: handle to cuSolver library
     hipblasFillMode_t: Indicates of matrix A lower or upper part stored
     int: dimension of matrix A
     float*: pointer to input matrix
     int: leading dimension of 2D array used to store matrix
     float*:workspace pointer
     int: size of workspace
     int*: return for error checking


  
  hipsolverDnDpotrf(csrHandle, uplo, 3, dA0, 3, r20work, r20workSize, devInfo);
  hipsolverDnDpotrf(csrHandle, uplo, r20n, r20Arr, r20n, 
                                      r20work, r20workSize, devInfo); 
  */
 
  //printf("Dev Info: %d", *devInfo);

//call function to perform cholesky
chol( dA0, 3, HIPBLAS_FILL_MODE_UPPER );   

   //fclose(fp);
  /* fp = fopen("test_corr_matrix_d=200.txt", "r"); 
   if(fp)
      {
        for(int i = 0; i < 200 * 200; i++)
           {
             fscanf(fp, "%f", &r200[i]);
	   }
      }*/

   //test input read by printing results
  printf("\n DECOMP RESULTS: \n");
  for(int i = 0; i < 3; i++ ){
    for(int j = 0; j <3; j++ )
      {
        printf(" %f", dA0[i*3+j]);
      } 
      printf("\n");
   }
//generate random variables matrix
size_t n = 10;
double * randMat;
hipMallocManaged( &randMat, 10*sizeof(double) );
normGen( randMat, n, 0.0,1.0 );

//print results to screen
printf("\n RANDOM MATRIX: \n");
for(int i = 0; i < 3; i++ ){
  for(int j=0; j < 3; j++){
    printf(" %f", randMat[i*3+j]);
    }
  printf("\n");
 }  

    /*for(int i = 0; i < 200; i++ ){
      for(int j = 0; j <200; j++ )
        {
          printf("%f", r200[i*20+j]);
        } 
        printf("\n");
   }  
     */ 
///////// generate random variable //////////////////////////////
//hiprandGenerateNormalDouble()
   //free memory
   hipFree(r20Arr);
   //hipFree(r20work);
   //hipFree(r200);
   hipFree(dA0);
   hipFree(randMat);   
}
