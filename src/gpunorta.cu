///////////////////////////////////////////////////////////////////////////////
///////////////   This is the GPU version of NORTA   //////////////////////////
///////////////////// Written by Eric Li //////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////



/////////////////////////////  Includes  //////////////////////////////////////
#include <cstdlib>
#include <fstream>
#include <cstdio>
#include <iostream>
#include <stdlib.h>
#include <hipsolver.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <string.h>

#include "cudaFuncs.h"
#include "stats.hpp"

using namespace std;


///////////////////////////  struct declaration  //////////////////////////////
//struct of arrays of distribution and their parameters
struct distStruct
   {
       int* distKey;
       float** params; 
   };

//////////////////////////// Function prototypes  /////////////////////////////
bool readFromFile(const char*, double*, int );
bool readDistFile( const char*, distStruct*, int );



///////////////////////////////////////////////////////////////////////////////
/////////////////////////////////    Main   ///////////////////////////////////
int main( int argc, char const *argv[])
{
 //initialize variables
   //initiliaze arrays for holding input data
   double* r20Arr;
   double* r20ArrNF; 
   double* r200Arr;
   double* r20501Arr;

   int r20n = 20;
   int r200n = 200;
   int r20501n = 20501;
   int n = 1093;
   int r20Size; 
   int r200Size;
   int r20501Size;
   int sim20501Size = n * r20501n;
   int d = 6;

   //ifstream srcFile;
   double A0[3*3] = { 1.0, 2.0, 3.0, 2.0, 5.0, 5.0, 3.0, 5.0, 12.0 };
   double AC[6] = {1.0, 2.0, 3.0, 5.0, 5.0, 12.0};
   double AR[6] = {1.0, 2.0, 5.0, 3.0, 5.0, 12.0};
   double testdata = 0.1;
   double testArr[6] = {0.1, 0.1, 0.1, 0.1, 0.1, 0.1 };
   double *dtestArr;
   double* dA0;
   double* dAR;
   double* dAC;
   double* sim_r20;
   double* sim_r200;
   double* sim_r20501;
   //file names
   char r20file[60] = "../test_corr_matrix_d=20.txt";
   char r200file[60] = "../test_corr_matrix_d=200.txt";
   char r20501file[60] = "../test_corr_matrix_d=20501.txt";
   char distFile[60] = "../distributions.txt";

   r20Size = r20n*r20n;
   r200Size = r200n*r200n;
   r20501Size = r20501n * r20501n;
   //initialize distribution struct for inverse prob
   distStruct dists;
   //initialize array for distributions
   hipMallocManaged( &(dists.distKey), 6*sizeof(int) );
   
   //initialize array of pointers for parameters.
   hipMallocManaged( &(dists.params), 6*sizeof(float*) );
   dists.params = new float*[ 6 ];

   //read in distributions file
   if( readDistFile( distFile, &dists, 6) ){
    cout << endl << "READ DIST FILE SUCCESS!";
   }

   /*ifstream src;
   char buffer[20];
   src.open(distFile, ifstream::in );
   for(int i = 0; i = 5; i++){
      src >> buffer;
      cout << buffer << ' ';
   }*/



   //cuSolver 
   //cuSolverStatus_t solverStatus;
   
   //hipStream_t stream = NULL;   
   
   //set stream
   //hipsolverSetStream(csrHandle, stream);

   /*//print cusolver version
   int major=-1,minor=-1,patch=-1;
    cusolverGetProperty(MAJOR_VERSION, &major);
    cusolverGetProperty(MINOR_VERSION, &minor);
    cusolverGetProperty(PATCH_LEVEL, &patch);
    printf("\n CUSOLVER Version (Major,Minor,PatchLevel): %d.%d.%d\n", major,minor,patch);*/

   //allocated unified memory for storage of input covar matrix. 
   hipMallocManaged(&r20Arr, r20Size*sizeof(double));
   //hipMallocManaged(&r20ArrNF, r20Size*sizeof(double));
   hipMallocManaged(&r200Arr, r200Size*sizeof(double));
   hipMallocManaged(&r20501Arr, r20501Size*sizeof(double));

   hipMallocManaged(&sim_r20, r20Size*sizeof(double));
   hipMallocManaged(&sim_r200, r200Size*sizeof(double));
   hipMallocManaged(&sim_r20501, sim20501Size*sizeof(double));

   //allocate device memory for simple testing
   hipMallocManaged( &dA0, 9*sizeof(double) );
   hipMallocManaged( &dAR, 6*sizeof(double) );
   hipMallocManaged( &dAC, 6*sizeof(double) );
   hipMallocManaged( &dtestArr, 6*sizeof(double) );

   //copy explicitly defined matrix into device
   hipMemcpy( dA0, A0, 9*sizeof(double), hipMemcpyHostToDevice );
   hipMemcpy( dAR, AR, 6*sizeof(double), hipMemcpyHostToDevice );
   hipMemcpy( dAC, AC, 6*sizeof(double), hipMemcpyHostToDevice );
   hipMemcpy( dtestArr, testArr, 6*sizeof(double), hipMemcpyHostToDevice );



   //hipMallocManaged(&r200Arr, r200Size*sizeof(float));
     
 //start timing
 
  //Timing for file read r20
   hipEvent_t readStart, readEnd;
   hipEventCreate( &readStart );
   hipEventCreate(  &readEnd );
   hipEventRecord( readStart, 0); 

 //call function to read in from file
 if( readFromFile( r20501file, r20501Arr, r20501Size) ){
   cout << endl << "FILE OPEN SUCCESS!";
 }  
 else{
   cout << endl << "ERROR FILE OPENING";
 }
 
 //stop timing
 hipEventRecord( readEnd, 0 );
 hipEventSynchronize( readEnd );
 float readTime;
 hipEventElapsedTime( &readTime, readStart, readEnd );
 //print timing results
 cout << endl << "Reading in r20501: " << readTime << " ms." << endl;

  /*//Timing for file read r200
   hipEventCreate( &readStart );
   hipEventCreate(  &readEnd );
   hipEventRecord( readStart, 0); 
*/
 /*//call function to read in from file
 if( readFromFile( r200file, r200Arr, r200Size) ){
   cout << endl << "FILE OPEN SUCCESS!";
 }  
 else{
   cout << endl << "ERROR FILE OPENING";
 }
 
 //stop timing
 hipEventRecord( readEnd, 0 );
 hipEventSynchronize( readEnd );
 float readTime;
 hipEventElapsedTime( &readTime, readStart, readEnd );
 */
 //print timing results
 //cout << endl << "Reading in r200: " << readTime << " ms." << endl;
 
/* //test input read by printing results
  printf("\n INITIAL MATRIX\n");
 
  for(int i = 0; i < 20; i++ ){
    for(int j = 0; j <20; j++ )
      {
        printf(" %f", r20Arr[i*3+j]);
      } 
      printf("\n");
   }
*/

  //printf("Dev Info: %d", *devInfo);
//Timing for cholesky r20
hipEvent_t cholStart, cholEnd;
hipEventCreate( &cholStart ); 
hipEventCreate( &cholEnd );
hipEventRecord( cholStart, 0 );

//call function to perform cholesky
chol( r20501Arr, r20501n, HIPBLAS_FILL_MODE_UPPER );   
//synchronize threads
hipDeviceSynchronize();
//chol(dAR, 3, HIPBLAS_FILL_MODE_LOWER );
//chol(dAC, 3, HIPBLAS_FILL_MODE_LOWER );
//End timing
hipEventRecord( cholEnd, 0);
hipEventSynchronize( cholEnd );
float cholTime;
hipEventElapsedTime( &cholTime, cholStart, cholEnd );
cout << endl << "Cholesky r20501 Took: " << cholTime << " ms." << endl;


//Timing for cholesky r200
//hipEventCreate( &cholStart ); 
//hipEventCreate( &cholEnd );
//hipEventRecord( cholStart, 0 );

/*//call function to perform cholesky
chol( r200Arr, 200, HIPBLAS_FILL_MODE_UPPER );   
//synchronize threads
hipDeviceSynchronize();

//End timing
hipEventRecord( cholEnd, 0);
hipEventSynchronize( cholEnd );
float cholTime1;
hipEventElapsedTime( &cholTime, cholStart, cholEnd );
cout << endl << "Cholesky r200 Took: " << cholTime1 << " ms." << endl;
*/
   //fclose(fp);
  /* fp = fopen("test_corr_matrix_d=200.txt", "r"); 
   if(fp)
      {
        for(int i = 0; i < 200 * 200; i++)
           {
             fscanf(fp, "%f", &r200[i]);
     }
      }*/

   /*//test input read by printing results
  printf("\n DECOMP RESULTS: \n");
  for(int i = 0; i < 3; i++ ){
    for(int j = 0; j <3; j++ )
      {
        printf(" %f", dA0[i*3+j]);
      } 
      printf("\n");
   }*/
///////// generate random variable //////////////////////////////
//size_t n = 10;
double * randMat;
int time1 = time(NULL);
hipMallocManaged( &randMat, 10*sizeof(double) );

//timing
hipEvent_t randStart, randEnd;
hipEventCreate( &randStart ); 
hipEventCreate( &randEnd );
hipEventRecord( randStart, 0 );

normGen( sim_r20501, sim20501Size, 0.0,1.0, time1 );

hipEventRecord( randEnd, 0);
hipEventSynchronize( randEnd );
float randTime;
hipEventElapsedTime( &randTime, randStart, randEnd );
cout << endl << "RNG r200: " << randTime << " ms." << endl;

cout <<endl << "TIME SEED: " << time1;
/*//print results to screen
printf("\n RANDOM MATRIX: \n");
for(int i = 0; i < 200; i++ ){
  for(int j=0; j < 200; j++){
    printf(" %f", sim_r200[i*3+j]);
    }
  printf("\n");
 }  */

    /*for(int i = 0; i < 200; i++ ){
      for(int j = 0; j <200; j++ )
        {
          printf("%f", r200[i*20+j]);
        } 
        printf("\n");
   }  
     */ 
/////////////////////  matrix multiplication  /////////////////////////////////
/*hipEvent_t multStart, multEnd;
hipEventCreate( &multStart );
hipEventCreate( &multEnd );
hipEventRecord( multStart, 0 );*/
double* M1;
double* M2;
double* M3;
//allocate memory for matrix testing
hipMallocManaged( &M1, 6*sizeof(double) );
hipMallocManaged( &M2, 3*sizeof(double) );
hipMallocManaged( &M3, 2*sizeof(double) );

for(int i = 0; i < 6; i ++){
  M1[i] = i;
}
for(int i = 0; i < 3; i++ ){
  M2[i] = i;
}

//parameters are: cols of M2, rows of M1, row of M2
matMult(M1, M2, M3, 1, 2, 3);

/*//print results
cout << endl << "MATRIX MULT RESULTS" << endl;
for(int i = 0; i < 2; i++ ){
  cout << endl << M3[i];
}
cout << endl;*/

 //multiplication of cholesky w/ random matrix to get correlated random matrix
 hipEvent_t multStart, multEnd;
 hipEventCreate( &multStart );
 hipEventCreate( &multEnd );
 hipEventRecord( multStart, 0 );
  
 matMult( sim_r20501, r20501Arr, sim_r20501, r20501n, n, r20501n );
 hipEventRecord( multEnd, 0);
 hipEventSynchronize( multEnd );
 float multTime;
 hipEventElapsedTime( &multTime, multStart, multEnd );
 cout << endl << "mult r20: " << multTime << " ms." << endl;
 
 /*//calling qnorm from stats lib works.
 cout << "TESTING FOR STATS LIBRARY" << endl;
 cout << "b4 Value = " << testdata;
 cout << "after value = " << stats::qnorm(testdata) << endl; 
*/

//testing to see if library can be called from kernel
/*cout << "TESTING KERNEL " << endl;
cout << "B4 ARRAY : ";
for(int i = 0; i < 6; i++ ){
  cout << dtestArr[i] << ' ';
}

cout << endl;
int ffff  = 1;
dim3 grid(1);
dim3 block(6);

hipDeviceSynchronize();
testFunc<<<grid,block>>>( dtestArr, 6 );
hipDeviceSynchronize();
cout << "AFTER ARRAY: ";
for(int i = 0; i < 6; i++){
  cout << dtestArr[i] << ' ';
}
cout << endl;
 */
   //free memory
   hipFree(r20Arr);
   hipFree(r200Arr);
   hipFree(r20501Arr);
   
   hipFree(dA0);
   hipFree(randMat); 
   hipFree(M1);
   hipFree(M2);
   hipFree(M3);
   hipFree(sim_r20);
   hipFree(sim_r200);
   hipFree(sim_r20501);
   hipFree(dists.distKey);
   
   for( int i = 0; i < d; i++ ){
    hipFree( dists.params[i] );
   }

   hipFree( dists.params );
}



/////////////////// Function Implementation ///////////////////////////////////
bool readFromFile( const char* fileName, double* output, int size ){
   ifstream source;
   source.open( fileName, fstream::in );

   if( source ){
      for( int i = 0; i < size; i++ )
         {
            source >> output[i];
         }
       source.close();  
       return true;
     }

   else{
     source.close();   
     return false;
     }

}

bool readDistFile(const char* fileName, distStruct* dists, int numDists ){
   ifstream source;
   char distName[20];
   source.open( fileName, fstream::in );
   float numBuffer;
   if( source ){
      //loop over all distributions 
      for( int i = 0; i < numDists; i++ ){ 
         source >> distName;
         cout << "NAME OF DIST: ";
         cout << endl << distName << endl;
         //test for each distribution supported, 14 total, 
         //sets params accordingly
         if( strcmp( "beta", distName) == 0 ){
            dists->distKey[i] = 0;
            hipMallocManaged( &( dists->params[i] ), 2*sizeof(float) );
            source >> dists->params[i][0];
            source >> dists->params[i][1];
         }

         /*else if( strcmp( "binomial", distName) == 0 ){
            dists->distKey[i] = 1;
            dists->params[i] = new float[2];
            source >> dists->params[i][0];
            source >> dists->params[i][1];
         }*/

         else if( strcmp( "cauchy", distName ) == 0 ){
            dists->distKey[i] = 2;
            hipMallocManaged( &( dists->params[i] ), 2*sizeof(float) );
            source >> dists->params[i][0];
            source >> dists->params[i][1];
         }

         else if( strcmp( "chi-squared", distName ) == 0 ){
            dists->distKey[i] = 3;
            hipMallocManaged( &( dists->params[i] ), 1*sizeof(float) );
            source >> dists->params[i][0];
         }

         else if( strcmp( "exponential", distName ) == 0 ){
            dists->distKey[i] = 4;
            hipMallocManaged( &( dists->params[i] ), 1*sizeof(float) );
            source >> dists->params[i][0];
         }

        else if( strcmp( "f", distName ) == 0 ){
            dists->distKey[i] = 5;
            hipMallocManaged( &( dists->params[i] ), 2*sizeof(float) );
            source >> dists->params[i][0];
            source >> dists->params[i][1];
         } 

        else if( strcmp( "gamma", distName ) == 0 ){
            dists->distKey[i] = 6;
            hipMallocManaged( &( dists->params[i] ), 2*sizeof(float) );
            source >> dists->params[i][0];
            source >> dists->params[i][1];
         } 

        else if( strcmp( "normal", distName ) == 0 ){
            dists->distKey[i] = 7;
            hipMallocManaged( &( dists->params[i] ), 2*sizeof(float) );
            source >> dists->params[i][0] >> dists->params[i][1];
            //source >> dists->params[i][1];
         } 

        else if( strcmp( "lognormal", distName ) == 0 ){
            dists->distKey[i] = 8;
            hipMallocManaged( &( dists->params[i] ), 2*sizeof(float) );
            source >> dists->params[i][0];
            source >> dists->params[i][1];
         }

        else if( strcmp( "logistic", distName ) == 0 ){
            dists->distKey[i] = 9;
            hipMallocManaged( &( dists->params[i] ), 2*sizeof(float) );
            source >> dists->params[i][0];
            source >> dists->params[i][1];
         }

        else if( strcmp( "poisson", distName ) == 0 ){
            dists->distKey[i] = 10;
            hipMallocManaged( &( dists->params[i] ), 1*sizeof(float) );
            source >> dists->params[i][0];
         }

        else if( strcmp( "t", distName ) == 0 ){
            dists->distKey[i] = 11;
            hipMallocManaged( &( dists->params[i] ), 1*sizeof(float) );
            source >> dists->params[i][0];
         }

        else if( strcmp( "uniform", distName ) == 0 ){
            dists->distKey[i] = 12;
            hipMallocManaged( &( dists->params[i] ), 2*sizeof(float) );
            source >> dists->params[i][0];
            source >> dists->params[i][1];
         }

        else if( strcmp( "weibull", distName ) == 0 ){
            dists->distKey[i] = 13;
            hipMallocManaged( &( dists->params[i] ), 2*sizeof(float) );
            source >> dists->params[i][0];
            source >> dists->params[i][1];
         } 

        else{
          cout << endl << "Error reading in paramters, check spelling.";
          return false;
        }               
      }
      source.close();
      return true;  
   }

   else{
      cout << endl << "Error opening distributions file";
      source.close();
      return false;
   }
}
